
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define RAD_CONV_FAC (1.0f/60.0f)*(M_PI/180.0f)
#define DEG_CONV_FAC 180.0f/M_PI

/*

Compile with: nvcc -O3 -Xptxas="-v" -arch=sm_30  galaxy_distribution.cu
Run with: time ./a.out real.txt sim.txt

*/

/* ---------- Device code ---------- */

/*   
     N = number of galaxies (in one list)
     a1 = ascension (alpha) for first galaxy list
     d1 = declination (delta) for first galaxy list
     a2 = ascension for second glaxy list
     d2 = declination for second galaxy list
*/
__global__ void theta_calc(int N, float* a1, float* d1, float* a2, float* d2, int* hist) {
     
     // Thread index
     int i = blockDim.x * blockIdx.x + threadIdx.x;

     if (i < N) {
          float angle, temp;
          int j, bin_nr;
          // Each thread multiplies one value (their index) of the first galaxy list against all the values in the second galaxy list
          for (j = 0; j < N; j++) {
               temp = sinf(d1[i])*sinf(d2[j])+cosf(d1[i])*cosf(d2[j])*cosf(a1[i]-a2[j]);
               angle = acosf(fminf(temp, 1.0f));
               // Convert to degree and determine bin number
               bin_nr = floor((angle*DEG_CONV_FAC)/0.25);
               // Increment histogram
               atomicAdd(&hist[bin_nr], 1);
          }
     }
}

void debug (int* dd, int* dr, int* rr, float* omega_hist, int bins) {

      // Save histograms to files for analysis
     FILE *out_file = fopen("output.txt", "w");
     FILE *out_file2 = fopen("output2.txt", "w");
     FILE *out_file3 = fopen("output3.txt", "w");
     FILE *out_file4 = fopen("output4.txt", "w");
     
     long int dd_tot = 0;
     long int dr_tot = 0;
     long int rr_tot = 0;

     for(int k = 0; k < bins; k++) {
          fprintf(out_file, "%d\n", dd[k]);
          fprintf(out_file2, "%d\n", dr[k]);
          fprintf(out_file3, "%d\n", rr[k]);
          fprintf(out_file4, "%f\n", omega_hist[k]);
          dd_tot += dd[k];
          dr_tot += dr[k];
          rr_tot += rr[k];
     }
     
     printf("Total entries in histogram dd: %ld\n", dd_tot);
     printf("Total entries in histogram dr: %ld\n", dr_tot);
     printf("Total entries in histogram rr: %ld\n", rr_tot);
}

void omega_calc(int* dd, int* dr, int* rr, float* omega_hist, int bins) {
     
     // Calculate omega (difference between two equally big sets) with the three histograms
     for (int m = 0; m < bins; m++) {
          omega_hist[m] = (float)((float)dd[m]-2.0f*(float)dr[m]+(float)rr[m])/(float)rr[m];
          if (m < 15) {
               printf("Omega %d: %f\n", m, omega_hist[m]);
          }
     }
}

/* ---------- Host code ---------- */

int main (int agrc, char *argv[]) {

     // Allocate arrays for the values
     int N = 100000;
     float* real_values_asc;
     float* real_values_dec;
     float* sim_values_asc;
     float* sim_values_dec;

     hipMallocManaged(&real_values_asc, N*sizeof(float)); 
     hipMallocManaged(&real_values_dec, N*sizeof(float));      
     hipMallocManaged(&sim_values_asc, N*sizeof(float));      
     hipMallocManaged(&sim_values_dec, N*sizeof(float));

     // Allocate arrays for the histograms
     int bins = 180*4;
     int* dd;
     int* dr;
     int* rr;
     float* omega_hist = (float*)malloc(bins*sizeof(float));

     hipMallocManaged(&dd, bins*sizeof(int));
     hipMallocManaged(&dr, bins*sizeof(int));
     hipMallocManaged(&rr, bins*sizeof(int));

     
     // Read values from the files
     FILE * file_real = fopen(argv[1], "r");
     FILE * file_sim = fopen(argv[2], "r");
     if(!file_real || !file_sim) {
          printf("Something went wrong with the file reading...\n");
          exit(-1);
     }
     
     for (int i = 0; i < 2*N; i++) { // The values are read one by one, not in pairs, so we need to double the iteration value
          if(i%2 == 0) { // First column, right ascension
               fscanf(file_real, "%f", &real_values_asc[i/2]);
               // Convert to radians
               real_values_asc[i/2] *= RAD_CONV_FAC;
               fscanf(file_sim, "%f", &sim_values_asc[i/2]);
               // Convert to radians
               sim_values_asc[i/2] *= RAD_CONV_FAC;
          }
          else { // Second column, declination
               fscanf(file_real, "%f", &real_values_dec[i/2]);
               // Convert to radians
               real_values_dec[i/2] *= RAD_CONV_FAC;
               fscanf(file_sim, "%f", &sim_values_dec[i/2]);
               // Convert to radians
               sim_values_dec[i/2] *= RAD_CONV_FAC;
          }

          // Initialize the histograms
          if(i < bins) {
               dd[i] = 0;
               dr[i] = 0;
               rr[i] = 0;
          }
     }
     
     fclose(file_real);
     fclose(file_sim);
     
     int threads_in_block = 512;
     int blocks_in_grid = (N+threads_in_block-1)/threads_in_block;

     // GPU function
     theta_calc<<<blocks_in_grid, threads_in_block>>>(N, real_values_asc, real_values_dec, real_values_asc, real_values_dec, dd);
     theta_calc<<<blocks_in_grid, threads_in_block>>>(N, real_values_asc, real_values_dec, sim_values_asc, sim_values_dec, dr);
     theta_calc<<<blocks_in_grid, threads_in_block>>>(N, sim_values_asc, sim_values_dec, sim_values_asc, sim_values_dec, rr);

     // Wait for GPU to finish
     hipDeviceSynchronize();

     // Calculate omega values
     omega_calc(dd, dr, rr, omega_hist, bins);
     
     //debug(dd, dr, rr, omega_hist, bins);

     // Free all the things
     hipFree(real_values_asc);
     hipFree(real_values_dec);
     hipFree(sim_values_asc);
     hipFree(sim_values_dec);
     hipFree(dd);
     hipFree(dr);
     hipFree(rr);
     free(omega_hist);
     
}